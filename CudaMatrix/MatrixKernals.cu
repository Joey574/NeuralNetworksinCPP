#include "hip/hip_runtime.h"
#include "MatrixKernals.cuh"

// Single Scalar operation

__global__ void add_scalar(float* matrix, float scalar, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] += scalar;
    }
}

__global__ void sub_scalar(float* matrix, float scalar, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] -= scalar;
    }
}

__global__ void mul_scalar(float* matrix, float scalar, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] *= scalar;
    }
}

__global__ void div_scalar(float* matrix, float scalar, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] /= scalar;
    }
}

__global__ void pow_scalar(float* matrix, float scalar, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] = powf(matrix[id], scalar);
    }
}

__global__ void exp_scalar(float* matrix, float scalar, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] = powf(scalar, matrix[id]);
    }
}

// Vector Operations

__global__ void add_vector(float* matrix, float* element, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] += element[id];
    }
}

__global__ void sub_vector(float* matrix, float* element, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] -= element[id];
    }
}

__global__ void mul_vector(float* matrix, float* element, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] *= element[id];
    }
}

__global__ void div_vector(float* matrix, float* element, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] /= element[id];
    }
}

__global__ void pow_vector(float* matrix, float* element, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] = powf(matrix[id], element[id]);
    }
}

__global__ void exp_vector(float* matrix, float* element, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] = powf(element[id], matrix[id]);
    }
}

// Uniform Math Operations

__global__ void sqrt(float* matrix, float scalar, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] = sqrtf(matrix[id]);
    }
}

__global__ void sin(float* matrix, float scalar, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] = sinf(matrix[id]);
    }
}

__global__ void cos(float* matrix, float scalar, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] = cosf(matrix[id]);
    }
}


__global__ void relu(float* matrix, float scalar, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] = fmaxf(matrix[id], 0.0f);
    }
}

__global__ void relu_derivative(float* matrix, float scalar, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        if (matrix[id] > 0.0f) {
            matrix[id] = 1.0f;
        } else {
            matrix[id] = 0.0f;
        }
    }
}


__global__ void leakyrelu(float* matrix, float scalar, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        matrix[id] = fmaxf(matrix[id], scalar * matrix[id]);
    }
}

__global__ void leakyrelu_derivative(float* matrix, float scalar, int num_elements) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_elements) {
        if (matrix[id] > 0.0f) {
            matrix[id] = 1.0f;
        }
        else {
            matrix[id] = scalar;
        }
    }
}